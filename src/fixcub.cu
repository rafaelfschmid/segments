/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <utility>
#include <iostream>
#include <bitset>
#include <math.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

void print(int* host_data, int n) {
	std::cout << "\n";
	for (int i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}

int main(void) {
	int num_of_segments;
	int num_of_elements;
	int i;

	scanf("%d", &num_of_segments);
	int mem_size_seg = sizeof(int) * (num_of_segments + 1);
	int *h_seg = (int *) malloc(mem_size_seg);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	int mem_size_vec = sizeof(int) * num_of_elements;
	int *h_vec = (int *) malloc(mem_size_vec);
	int *h_value = (int *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++) {
		scanf("%d", &h_vec[i]);
		h_value[i] = i;
	}

	int *h_norm = (int *) malloc(mem_size_seg);
	int previousMax = 0;
	for (i = 0; i < num_of_segments; i++) {
		int currentMin = h_vec[h_seg[i]];
		int currentMax = h_vec[h_seg[i]];

		for (int j = h_seg[i] + 1; j < h_seg[i + 1]; j++) {
			if (h_vec[j] < currentMin)
				currentMin = h_vec[j];
			else if (h_vec[j] > currentMax)
				currentMax = h_vec[j];
		}

		int normalize = previousMax - currentMin;
		h_norm[i] = ++normalize;
		for (int j = h_seg[i]; j < h_seg[i + 1]; j++) {
			h_vec[j] += normalize;
		}
		previousMax = currentMax + normalize;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *d_value, *d_value_out, *d_vec, *d_vec_out;
	void *d_temp = NULL;
	size_t temp_bytes = 0;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_vec_out, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value_out, mem_size_vec));

	for (int i = 0; i < EXECUTIONS; i++) {

		cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));
		cudaTest(hipMemcpy(d_value, h_value, mem_size_vec, hipMemcpyHostToDevice));

		if(temp_bytes == 0) {
			hipcub::DeviceRadixSort::SortPairs(d_temp, temp_bytes, d_vec, d_vec_out,
					d_value, d_value_out, num_of_elements);
			hipMalloc((void **) &d_temp, temp_bytes);
		}
		hipEventRecord(start);
		hipcub::DeviceRadixSort::SortPairs(d_temp, temp_bytes, d_vec, d_vec_out,
				d_value, d_value_out, num_of_elements);
		hipEventRecord(stop);

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	hipMemcpy(h_vec, d_vec_out, mem_size_vec, hipMemcpyDeviceToHost);

	for (i = 0; i < num_of_segments; i++) {
		for (int j = h_seg[i]; j < h_seg[i + 1]; j++) {
			h_vec[j] -= h_norm[i];
		}
	}

	hipFree(d_vec);
	hipFree(d_vec_out);
	hipFree(d_value);
	hipFree(d_value_out);
	hipFree(d_temp);

	if (ELAPSED_TIME != 1) {
		print(h_vec, num_of_elements);
	}

	free(h_seg);
	free(h_vec);
	free(h_norm);
	free(h_value);

	return 0;
}
