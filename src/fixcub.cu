#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_reduce.cuh>

#include <stdio.h>
#include <stdlib.h>
#include <bitset>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>

typedef unsigned int uint;

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

void print(uint* host_data, uint n) {
	std::cout << "\n";
	for (uint i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}

int main(void) {
	uint num_of_segments;
	uint num_of_elements;
	uint i;

	scanf("%d", &num_of_segments);
	uint mem_size_seg = sizeof(uint) * (num_of_segments + 1);
	uint *h_seg = (uint *) malloc(mem_size_seg);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	int mem_size_vec = sizeof(uint) * num_of_elements;
	uint *h_vec = (uint *) malloc(mem_size_vec);
	uint *h_value = (uint *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++) {
		scanf("%d", &h_vec[i]);
		h_value[i] = i;
	}

	/*
	 * maximum element of the array.
	 */
	uint maxValue = 0;
	for (i = 0; i < num_of_elements; i++) {
		if(maxValue < h_vec[i])
			maxValue = h_vec[i];
	}

	printf("max_val V1=%d", maxValue);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_value, *d_value_out, *d_vec, *d_vec_out, *d_max;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_max, sizeof(uint)));

	void *d_temp_storage = NULL;
	size_t temp_storage_bytes = 0;
	hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_vec, d_max, num_of_elements);
	// Allocate temporary storage
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	// Run max-reduction
	hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_vec, d_max, num_of_elements);

	int* max_val;
	*max_val = 0;
	hipMemcpy(max_val, d_max, (int)sizeof(uint), hipMemcpyDeviceToHost);
	printf("max_val V2=%d", max_val);

	/*uint mostSignificantBit = (uint)log2((double)max_val) + 1;

	for (i = 0; i < num_of_segments; i++) {
		for (uint j = h_seg[i]; j < h_seg[i + 1]; j++) {
			uint segIndex = i << mostSignificantBit;
			h_vec[j] += segIndex;
		}
	}

	void *d_temp = NULL;
	size_t temp_bytes = 0;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_vec_out, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value_out, mem_size_vec));

	for (uint i = 0; i < EXECUTIONS; i++) {

		cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));
		cudaTest(hipMemcpy(d_value, h_value, mem_size_vec, hipMemcpyHostToDevice));

		if(temp_bytes == 0) {
			hipcub::DeviceRadixSort::SortPairs(d_temp, temp_bytes, d_vec, d_vec_out,
					d_value, d_value_out, num_of_elements);
			hipMalloc((void **) &d_temp, temp_bytes);
		}
		hipEventRecord(start);
		hipcub::DeviceRadixSort::SortPairs(d_temp, temp_bytes, d_vec, d_vec_out,
				d_value, d_value_out, num_of_elements);
		hipEventRecord(stop);

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	hipMemcpy(h_vec, d_vec_out, mem_size_vec, hipMemcpyDeviceToHost);

	for (i = 0; i < num_of_segments; i++) {
		for (uint j = h_seg[i]; j < h_seg[i + 1]; j++) {
			uint segIndex = i << mostSignificantBit;
			h_vec[j] -= segIndex;
		}
	}*/

	hipFree(d_temp_storage);
	hipFree(d_max);
	hipFree(d_vec);
	/*hipFree(d_vec_out);
	hipFree(d_value);
	hipFree(d_value_out);
	hipFree(d_temp);*/

	if (ELAPSED_TIME != 1) {
		print(h_vec, num_of_elements);
	}

	free(h_seg);
	free(h_vec);
	free(h_value);

	return 0;
}
