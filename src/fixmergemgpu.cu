/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <moderngpu/kernel_mergesort.hxx>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <utility>
#include <iostream>
#include <bitset>
#include <math.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>
//#include <cstdlib>
#include <iostream>
#include <chrono>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

#ifndef EXECUTIONS
#define EXECUTIONS 10
#endif

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

void print(uint* host_data, uint n) {
	std::cout << "\n";
	for (uint i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}
////////////////////////////////////////////////////////////////////////////////
// Test driver
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
	uint num_of_segments;
	uint num_of_elements;
	uint i;

	scanf("%d", &num_of_segments);
	uint mem_size_seg = sizeof(int) * (num_of_segments + 1);
	uint *h_seg = (uint *) malloc(mem_size_seg);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	uint mem_size_vec = sizeof(int) * num_of_elements;
	uint *h_vec = (uint *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++) {
		scanf("%d", &h_vec[i]);
	}

	uint *h_norm = (uint *) malloc(mem_size_seg);
	uint previousMax = 0;
	for (i = 0; i < num_of_segments; i++) {
		uint currentMin = h_vec[h_seg[i]];
		uint currentMax = h_vec[h_seg[i]];

		for (uint j = h_seg[i] + 1; j < h_seg[i + 1]; j++) {
			if (h_vec[j] < currentMin)
				currentMin = h_vec[j];
			else if (h_vec[j] > currentMax)
				currentMax = h_vec[j];
		}

		uint normalize = previousMax - currentMin;
		h_norm[i] = ++normalize;
		for (uint j = h_seg[i]; j < h_seg[i + 1]; j++) {
			h_vec[j] += normalize;
		}
		previousMax = currentMax + normalize;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	uint *d_vec;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));

	for (int i = 0; i < EXECUTIONS; i++) {

		cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));

		hipEventRecord(start);
		mgpu::standard_context_t context;
		mgpu::mergesort(d_vec, num_of_elements, mgpu::less_t<int>(), context);
		hipEventRecord(stop);

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		if (ELAPSED_TIME == 1) {
			hipEventSynchronize(stop);
			float milliseconds = 0;
			hipEventElapsedTime(&milliseconds, start, stop);
			std::cout << milliseconds << "\n";
		}

		hipDeviceSynchronize();
	}

	hipMemcpy(h_vec, d_vec, mem_size_vec, hipMemcpyDeviceToHost);

	for (i = 0; i < num_of_segments; i++) {
		for (int j = h_seg[i]; j < h_seg[i + 1]; j++) {
			h_vec[j] -= h_norm[i];
		}
	}

	hipFree(d_vec);

	if (ELAPSED_TIME != 1) {
		print(h_vec, num_of_elements);
	}

	free(h_seg);
	free(h_vec);
	free(h_norm);
}
