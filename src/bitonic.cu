#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#include <stdio.h>
#include <stdlib.h>

#include <cutil.h>

#define NUM    256

__device__ inline void swap(int & a, int & b)
{
	// Alternative swap doesn't use a temporary register:
	// a ^= b;
	// b ^= a;
	// a ^= b;

    int tmp = a;
    a = b;
    b = tmp;
}

__global__ static void bitonicSort(int * values)
{
    extern __shared__ int shared[];

    const int tid = threadIdx.x;

    // Copy input to shared mem.
    shared[tid] = values[tid];

    __syncthreads();

    // Parallel bitonic sort.
    for (int k = 2; k <= NUM; k *= 2)
    {
        // Bitonic merge:
        for (int j = k / 2; j>0; j /= 2)
        {
            int ixj = tid ^ j;

            if (ixj > tid)
            {
                if ((tid & k) == 0)
                {
                    if (shared[tid] > shared[ixj])
                    {
                        swap(shared[tid], shared[ixj]);
                    }
                }
                else
                {
                    if (shared[tid] < shared[ixj])
                    {
                        swap(shared[tid], shared[ixj]);
                    }
                }
            }

            __syncthreads();
        }
    }

    // Write result.
    values[tid] = shared[tid];
}
//
// A sorting network is a sorting algorith, where the sequence of comparisons
// is not data-dependent. That makes them suitable for parallel implementations.
//
// Bitonic sort is one of the fastest sorting networks, consisting of o(n log^2 n)
// comparators. It has a simple implemention and it's very efficient when sorting 
// a small number of elements:
//
// http://citeseer.ist.psu.edu/blelloch98experimental.html
//
// This implementation is based on:
//
// http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
//

int main(int argc, char** argv)
{
    CUT_DEVICE_INIT();

    int values[NUM];

    for(int i = 0; i < NUM; i++)
    {
        values[i] = rand();
    }

    int * dvalues;
    CUDA_SAFE_CALL(hipMalloc((void**)&dvalues, sizeof(int) * NUM));
    CUDA_SAFE_CALL(hipMemcpy(dvalues, values, sizeof(int) * NUM, hipMemcpyHostToDevice));

    bitonicSort<<<1, NUM, sizeof(int) * NUM>>>(dvalues);

    // check for any errors
    CUT_CHECK_ERROR("Kernel execution failed");

    CUDA_SAFE_CALL(hipMemcpy(values, dvalues, sizeof(int) * NUM, hipMemcpyDeviceToHost));

    CUDA_SAFE_CALL(hipFree(dvalues));

    bool passed = true;
    for(int i = 1; i < NUM; i++)
    {
        if (values[i-1] > values[i])
        {
            passed = false;
        }
    }

    printf( "Test %s\n", passed ? "PASSED" : "FAILED");

    CUT_EXIT(argc, argv);
}
