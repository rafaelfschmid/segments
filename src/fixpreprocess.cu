/*
 ============================================================================
 Name        : sorting_segments.cu
 Author      : Rafael Schmid
 Version     :
 Copyright   : Your copyright notice
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>

#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <utility>
#include <iostream>
#include <bitset>
#include <math.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#ifndef ELAPSED_TIME
#define ELAPSED_TIME 0
#endif

void cudaTest(hipError_t error) {
	if (error != hipSuccess) {
		printf("cuda returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit (EXIT_FAILURE);
	}
}

void print(int* host_data, int n) {
	std::cout << "\n";
	for (int i = 0; i < n; i++) {
		std::cout << host_data[i] << " ";
	}
	std::cout << "\n";
}

int main(void) {
	int num_of_segments;
	int num_of_elements;
	int i;

	scanf("%d", &num_of_segments);
	int mem_size_seg = sizeof(int) * (num_of_segments + 1);
	int *h_seg = (int *) malloc(mem_size_seg);
	for (i = 0; i < num_of_segments + 1; i++)
		scanf("%d", &h_seg[i]);

	scanf("%d", &num_of_elements);
	int mem_size_vec = sizeof(int) * num_of_elements;
	int *h_vec_aux = (int *) malloc(mem_size_vec);
	int *h_value = (int *) malloc(mem_size_vec);
	for (i = 0; i < num_of_elements; i++) {
		scanf("%d", &h_vec_aux[i]);
		h_value[i] = i;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *d_value, *d_value_out, *d_vec, *d_vec_out;
	void *d_temp = NULL;
	size_t temp_bytes = 0;

	cudaTest(hipMalloc((void **) &d_vec, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_vec_out, mem_size_vec));
	cudaTest(hipMalloc((void **) &d_value_out, mem_size_vec));

	int *h_vec = (int *) malloc(mem_size_vec);
	int *h_norm = (int *) malloc(mem_size_seg);
	for (int k = 0; k < EXECUTIONS; k++) {

		for(int j = 0; j < num_of_elements; j++)
			h_vec[j] = h_vec_aux[j];

		std::chrono::high_resolution_clock::time_point start1 =
				std::chrono::high_resolution_clock::now();
		int previousMax = 0;
		for (i = 0; i < num_of_segments; i++) {
			int currentMin = h_vec[h_seg[i]];
			int currentMax = h_vec[h_seg[i]];

			for (int j = h_seg[i] + 1; j < h_seg[i + 1]; j++) {
				if (h_vec[j] < currentMin)
					currentMin = h_vec[j];
				else if (h_vec[j] > currentMax)
					currentMax = h_vec[j];
			}

			int normalize = previousMax - currentMin;
			h_norm[i] = ++normalize;
			for (int j = h_seg[i]; j < h_seg[i + 1]; j++) {
				h_vec[j] += normalize;
			}
			previousMax = currentMax + normalize;
		}
		std::chrono::high_resolution_clock::time_point stop1 =
				std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> time_span = std::chrono::duration_cast<
				std::chrono::duration<double>>(stop1 - start1);

		cudaTest(hipMemcpy(d_vec, h_vec, mem_size_vec, hipMemcpyHostToDevice));
		cudaTest(hipMemcpy(d_value, h_value, mem_size_vec,	hipMemcpyHostToDevice));

		if(temp_bytes == 0) {
			hipcub::DeviceRadixSort::SortPairs(d_temp, temp_bytes, d_vec, d_vec_out,
					d_value, d_value_out, num_of_elements);
			hipMalloc((void **) &d_temp, temp_bytes);
		}
		hipcub::DeviceRadixSort::SortPairs(d_temp, temp_bytes, d_vec, d_vec_out,
				d_value, d_value_out, num_of_elements);

		hipError_t errSync = hipGetLastError();
		hipError_t errAsync = hipDeviceSynchronize();
		if (errSync != hipSuccess)
			printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
		if (errAsync != hipSuccess)
			printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

		cudaTest(hipMemcpy(h_vec, d_vec_out, mem_size_vec, hipMemcpyDeviceToHost));

		start1 = std::chrono::high_resolution_clock::now();
		for (i = 0; i < num_of_segments; i++) {
			for (int j = h_seg[i]; j < h_seg[i + 1]; j++) {
				h_vec[j] -= h_norm[i];
			}
		}
		stop1 = std::chrono::high_resolution_clock::now();
		time_span += std::chrono::duration_cast<std::chrono::duration<double>>(
				stop1 - start1);

		if (ELAPSED_TIME == 1) {
			std::cout << time_span.count()*1000 << "\n";
		}

		hipDeviceSynchronize();
	}

	hipFree (d_vec);
	hipFree (d_vec_out);
	hipFree (d_value);
	hipFree (d_value_out);
	hipFree (d_temp);


	if (ELAPSED_TIME != 1) {
		print(h_vec, num_of_elements);
	}

	free(h_seg);
	free(h_vec);
	free(h_norm);
	free(h_vec_aux);
	free(h_value);

	return 0;
}
